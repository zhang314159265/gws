#include "hip/hip_runtime.h"
/*
 * Code copied from llm.c revision f1e2ace6 (May 10 2025) with the following changes worth noting
 * - Use '=' rather than '+=' to compute dx/dw/db
 * - pass in an argument BT (as B * T) rather than passing in B & T separately
 */
#include <hip/hip_bf16.h>

typedef __hip_bfloat16 floatX;

#define WARP_SIZE 32U

#define CEIL_DIV(M, N) (((M) + (N)-1) / (N))

template<class ElementType>
struct alignas(16) Packed128 {
    Packed128() = default;
    __device__ explicit Packed128(int4 bits) {
        static_assert(sizeof(bits) == sizeof(payload), "Size mismatch.");
        memcpy(&payload, &bits, sizeof(bits));
    }

    __device__  static Packed128 constant(ElementType value) {
        Packed128 result;
        for(int k = 0; k < size; ++k) {
            result.payload[k] = value;
        }
        return result;
    }
    __device__ static Packed128 zeros() {
        return constant(0.f);
    }
    __device__ static Packed128 ones() {
        return constant(1.f);
    }

    __device__ ElementType& operator[](int index) {
        return payload[index];
    }
    __device__ const ElementType& operator[](int index) const {
        return payload[index];
    }
    __device__ int4 get_bits() const {
        int4 bits;
        static_assert(sizeof(bits) == sizeof(payload), "Size mismatch.");
        memcpy(&bits, &payload, sizeof(bits));
        return bits;
    }
    static constexpr const size_t size = sizeof(int4) / sizeof(ElementType);
    ElementType payload[size];
};

// load a Packed128 from an aligned memory address
template<class ElementType>
__device__ Packed128<ElementType> load128(const ElementType* address) {
    return Packed128<ElementType>{*reinterpret_cast<const int4*>(address)};
}
// load a Packed128 from an aligned memory address with streaming cache hint
template<class ElementType>
__device__ Packed128<ElementType> load128cs(const ElementType* address) {
    return Packed128<ElementType>{__ldcs(reinterpret_cast<const int4*>(address))};
}
// store a Packed128 to an aligned memory address
template<class ElementType>
__device__ void store128(ElementType* target, Packed128<ElementType> value) {
    *reinterpret_cast<int4*>(target) = value.get_bits();
}
// store a Packed128 to an aligned memory address with streaming cache hint
template<class ElementType>
__device__ void store128cs(ElementType* target, Packed128<ElementType> value) {
    __stcs(reinterpret_cast<int4*>(target), value.get_bits());
}
// store a Packed128 to an aligned memory address while caching in L2 but bypassing L1
template<class ElementType>
__device__ void store128cg(ElementType* target, Packed128<ElementType> value) {
    __stcg(reinterpret_cast<int4*>(target), value.get_bits());
}

// short-form typedefs
typedef Packed128<float> f128;
typedef Packed128<floatX> x128;

__device__ inline float warpReduceSum(float val) {
    for (int offset = 16; offset > 0; offset /= 2) {
        val += __shfl_xor_sync(0xFFFFFFFF, val, offset);
    }
    return val;
}

extern "C" __global__ void __launch_bounds__(512, 2) // todo - any warnings on Turing with only 1024 threads?
    layernorm_bwd_kernel(floatX* dinp, floatX* dweight, floatX* dbias, float* scratch,
                                const floatX* dout, const floatX* inp, const floatX* weight,
                                const float* mean, const float* rstd,
                                int BT, int C) {
    int BLOCK_SIZE = blockDim.x;
    int warpsInBlock = BLOCK_SIZE / WARP_SIZE; //number of warps in block
    extern __shared__ float shared[];

    int warpId = threadIdx.x / WARP_SIZE; // warp index within a block
    int baseIdx = blockIdx.x * warpsInBlock + warpId;
    int warpThreadIdx = threadIdx.x % WARP_SIZE; // Thread index within the warp
    int warpsInGrid = gridDim.x * warpsInBlock;
    int C_per_iteration = WARP_SIZE * x128::size;
    int iterations_C = CEIL_DIV(C, C_per_iteration); // + 2;

    // the first half of shared memory is bias, second is weight
    size_t rounded_C = CEIL_DIV(C, (32 * x128::size)) * (32 * x128::size);
    float* dbias_shared = shared;
    float* dweight_shared = shared + rounded_C;
    // warp zero doesn't actually write to the _tmp_shared memory locations, so we don't need to reserve memory
    // the obvious solution is to change the addressing below to use (threadId.x-32) as offset, but that causes
    // register spills, so instead we mess with the base pointer here, which doesn't increase register usage.
    float* dbias_tmp_shared = shared + 2 * rounded_C - WARP_SIZE * f128::size;
    float* dweight_tmp_shared = shared + 2 * rounded_C + f128::size * BLOCK_SIZE - 2 * WARP_SIZE * f128::size;

    // init shared memory to zero
    for(int i = threadIdx.x * f128::size; i < rounded_C; i += BLOCK_SIZE * f128::size) {
        store128(dbias_shared + i, f128::zeros());
        store128(dweight_shared + i, f128::zeros());
    }
    __syncthreads();

    for (int bt = baseIdx; bt < BT; bt += warpsInGrid) {
        const floatX* dout_bt = dout + bt * C;
        const floatX* inp_bt = inp +bt * C;
        floatX* dinp_bt = dinp + bt * C;

        // first: two reduce operations
        float dnorm_mean = 0.0f;
        float dnorm_norm_mean = 0.0f;
        for (int i = warpThreadIdx * x128::size; i < C; i += WARP_SIZE * x128::size) {
            x128 dout128_i   = load128(dout_bt + i);
            x128 inp128_i    = load128(inp_bt  + i);
            x128 weight128_i = load128(weight  + i);
            for (int k = 0; k < x128::size; k++) {
                float dnorm_i = (float)weight128_i[k] * (float)dout128_i[k];
                dnorm_mean += dnorm_i;
                dnorm_norm_mean += dnorm_i * (float)inp128_i[k];
            }
        }

        const float mean_bt = mean[bt];
        const float rstd_bt = rstd[bt];
        dnorm_mean = warpReduceSum(dnorm_mean) / C;
        dnorm_norm_mean = warpReduceSum(dnorm_norm_mean) / C * rstd_bt - dnorm_mean * mean_bt * rstd_bt;

        for (int c = 0; c < iterations_C; c++) {
            int global_index = (warpThreadIdx * x128::size) + (c * C_per_iteration);

            x128 dout128   = x128::zeros();
            x128 inp128    = x128::zeros();
            x128 dinp128   = x128::zeros();
            x128 weight128 = x128::zeros();

            if(global_index < C) {
                dout128 = load128cs(dout_bt + global_index);
                inp128 = load128cs(inp_bt + global_index);
                // dinp128 = load128(dinp_bt + global_index);
                weight128 = load128(weight + global_index);
            }

            for(int o = 0; o < x128::size / f128::size; ++o) {
                f128 dbias_f;
                f128 dweight_f;
                for(int i = 0; i < f128::size; ++i) {
                    int x = o * f128::size + i;
                    float dout_i = (float)dout128[x];
                    float norm_bti = ((float)inp128[x] - mean_bt) * rstd_bt;
                    dbias_f[i] = dout_i;
                    dweight_f[i] = norm_bti * dout_i;

                    float dval = 0.0f;
                    dval += (float) weight128[x] * (float)dout128[x]; // term 1
                    dval -= dnorm_mean; // term 2
                    dval -= norm_bti * dnorm_norm_mean; // term 3
                    dval *= rstd_bt; // final scale
                    dinp128[x] = (floatX) ((float) dval);
                }

                if (warpId != 0) {
                    store128(dbias_tmp_shared + threadIdx.x * f128::size, dbias_f);
                    // this seems to generate a 64-bit store, instead of 128-bit.
                    // however, forcing 128-bit (e.g., using inline ptx), results in register
                    // spilling and much worse performance, so we'll keep it like this for now
                    // but ideally, we could reduce the register pressure a little.
                    store128(dweight_tmp_shared + threadIdx.x * f128::size, dweight_f);
                }
                __syncthreads();
                if (warpId == 0) {
                    for (int j = 1; j < warpsInBlock; j++) {
                        f128 dbias_tmp = load128(dbias_tmp_shared + f128::size * (threadIdx.x + j * WARP_SIZE));
                        f128 dweight_tmp = load128(dweight_tmp_shared + f128::size * (threadIdx.x + j * WARP_SIZE));
                        for(int i = 0; i < f128::size; ++i) {
                            dbias_f[i] += dbias_tmp[i];
                            dweight_f[i] += dweight_tmp[i];
                        }
                    }
                }
                __syncthreads();
                if (warpId == 0) {
                    f128 db_old = load128(dbias_shared + global_index + f128::size * o);
                    f128 dw_old = load128(dweight_shared + global_index + f128::size * o);
                    for(int i = 0; i < f128::size; ++i) {
                        dbias_f[i] += db_old[i];
                        dweight_f[i] += dw_old[i];
                    }
                    store128(dbias_shared + global_index + f128::size * o, dbias_f);
                    store128(dweight_shared + global_index + f128::size * o, dweight_f);
                }
            }
            if(global_index < C) {
                // cache in L2 as this is read by the next kernel, but bypass L1 to minimise thrashing
                store128cg(dinp_bt + global_index, dinp128);
            }
        }
    }
    __syncthreads();
    // Each block writes its partial sum to global memory
    // The last block to finish becomes responsible for summing up all the partial sums
    // This is done by atomically incrementing a flag (cleared to 0 before launching the kernel)
    unsigned int* scratchFlag = (unsigned int*)(scratch);
    // Increment scratch pointer by a full cacheline so that everything remains cacheline aligned
    scratch += 32;
    float* scratch_dbias = scratch;
    float* scratch_dweight = scratch + C;
    for(int i = threadIdx.x * f128::size; i < C; i += BLOCK_SIZE * f128::size) {
        // Write to global memory in the same "shared memory banking friendly" order
        store128(scratch_dbias + i + 2*C*blockIdx.x, load128(dbias_shared + i));
        store128(scratch_dweight + i + 2*C*blockIdx.x, load128(dweight_shared + i));
    }
    __syncthreads();
    // that portion of shared memory is no longer used, so we can repurpose it for the scratch flag.
    unsigned int *tmp_flag = (unsigned int*)(shared + 2*rounded_C);
    if (threadIdx.x == 0) {
        *tmp_flag = atomicInc(scratchFlag, gridDim.x);
    }
    __syncthreads();
    if (*tmp_flag == gridDim.x-1) {
        // Reduction of the partial sums by the final block
        // todo - there isn't enough parallelism even inside that single SM...
        // ==> so could maybe split into another kernel with YET ANOTHER level of reduction?!
        for(int i = threadIdx.x * f128::size; i < C; i += BLOCK_SIZE * f128::size) {
            f128 dbias_accum = f128::zeros();
            f128 dweight_accum = f128::zeros();

            for (int read_block_idx = 0; read_block_idx < gridDim.x; read_block_idx++) {
                int offset = i + 2*C*read_block_idx;
                f128 dbias128 = load128(scratch_dbias + offset);
                f128 dweight128 = load128(scratch_dweight + offset);
                for(int k = 0; k < f128::size; k++) {
                    dbias_accum[k] += dbias128[k];
                    dweight_accum[k] += dweight128[k];
                }
            }
            store128(dbias_shared + i, dbias_accum);
            store128(dweight_shared + i, dweight_accum);
        }
        __syncthreads();

        // convert from float/FP32 to floatX/BF16 for the final write
        // this is separate because it cannot use as many warps as the above (f128 vs x128)
        // todo - if we split this code into another kernel, we could maybe do it at the same time?
        for (int c = warpId; c < iterations_C; c += warpsInBlock) {
            int global_index = (warpThreadIdx * x128::size) + (c * C_per_iteration);
            if (global_index >= C) {
                break;
            }

            x128 dbias128; //  = load128(dbias + global_index);
            x128 dweight128; // = load128(dweight + global_index);
            for(int o = 0; o < x128::size / f128::size; ++o) {
                f128 s_db = load128(dbias_shared + global_index + o * f128::size);
                f128 s_dw = load128(dweight_shared + global_index + o * f128::size);
                for(int i = 0; i < f128::size; ++i) {
                    int x = o * f128::size + i;
                    dbias128[x] = (floatX)(s_db[i]);
                    dweight128[x] = (floatX)(s_dw[i]);
                }
            }
            store128(dbias + global_index, dbias128);
            store128(dweight + global_index, dweight128);
        }
    }
}
